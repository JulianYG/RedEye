#include "hip/hip_runtime.h"
#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/energy_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
void EnergyLossLayer<Dtype>::LayerSetUp(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {
	// set default loss weight 0.4
	if (this->layer_param_.loss_weight_size() == 0) {
    	this->layer_param_.add_loss_weight(Dtype(0.4));
  	}
  	_alpha = this->layer_param_.energy_loss_param().alpha();
	_model = this->layer_param_.energy_loss_param().model();
	_param_types = vector<int>(bottom.size(), 0);
	// initialize the vector indicating noise type
	if (this->layer_param_.energy_loss_param().paramtype_size())
		for (int i = 0; i < bottom.size(); i++)
			_param_types[i] = this->layer_param_.energy_loss_param().paramtype(i);
}

template <typename Dtype>
void EnergyLossLayer<Dtype>::Reshape(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {
	
	vector<int> top_shape(0);
	top[0]->Reshape(top_shape);
	// initialize the loss value
	top[0]->mutable_gpu_data()[0] = (Dtype) 0.;
	LOG(INFO) << "Scaling parameter value: " << _alpha;
}

template <typename Dtype>
void EnergyLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {
	for (int bottom_id = 0; bottom_id < bottom.size(); bottom_id++) {
		const Dtype* noise_param = bottom[bottom_id]->gpu_data();
		const int param_size = bottom[bottom_id]->count();
		LOG(INFO) << "Noise parameters blob length: " << param_size;
		Dtype loss_val = (Dtype) 0.;
		switch (_param_types[bottom_id]) {
			case 0: {	// case for gaussian, default
				for (int i = 0; i < param_size; i++) {
					switch(_model) {
						case EnergyLossParameter_ModelType_LINEAR: {
							loss_val -= _alpha * noise_param[i];
						} 	break;
					    case EnergyLossParameter_ModelType_SQUARE_INVERSE: { 
					    	loss_val += _alpha / pow(noise_param[i], 2);
						} 	break;
						case EnergyLossParameter_ModelType_RESERVED: {}	break;
					}
				}
			}	break;
			case 1: {}	break;	// case for poisson, no energy loss
			case 2: {	// case for uniform additive noise
				for (int i = 0; i < param_size; i++) {
					switch(_model) {
						case EnergyLossParameter_ModelType_LINEAR: {
							loss_val -= _alpha * noise_param[i];
						} 	break;
					    case EnergyLossParameter_ModelType_SQUARE_INVERSE: { 
					    	loss_val += _alpha / pow(noise_param[i], 2);
						} 	break;
						case EnergyLossParameter_ModelType_RESERVED: {}	break;
					}
				}
			} 	break;
		}	// also needs normalization
		top[0]->mutable_gpu_data()[0] += loss_val / param_size;
		LOG(INFO) << "Energy loss value for this bottom blob: " << loss_val / param_size;
	}
	LOG(INFO) << "top loss output: " << top[0]->mutable_gpu_data()[0];
	LOG(INFO) << "example noise param: " << bottom[0]->gpu_data()[0];
}

template <typename Dtype>
void EnergyLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

}

INSTANTIATE_LAYER_GPU_FUNCS(EnergyLossLayer);

}	// namespace caffe
