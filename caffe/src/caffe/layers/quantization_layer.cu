#include "hip/hip_runtime.h"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/layers/quantization_layer.hpp"

namespace caffe {

template<typename Dtype>
void QuantizationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
	const vector<Blob<Dtype>*>& top) {

	const Dtype* bottom_data = bottom[0]->cpu_data();
	Dtype* top_data = top[0]->mutable_cpu_data();
	
	Dtype maxVal = bottom_data[0];
	Dtype minVal = bottom_data[0];

	for (int i = 0; i < bottom[0]->count(); ++i) {
		if (minVal > bottom_data[i]) {
			minVal = bottom_data[i];
		} else if (maxVal < bottom_data[i]) {
			maxVal = bottom_data[i];
		}
	}
	Dtype bin = (maxVal - minVal) / Dtype(pow(2, _bit_num) - 1.);
	
	// for (int i = 0; i < top[0]->count(); ++i) {
	// 	top_data[i] = bin * floor((bottom_data[i] - minVal)/bin) + minVal;
	// }

	// hipblasHandle_t handle;
	// hipblasCreate(&handle);
	// int minIdx, maxIdx;
	// hipblasIsamax(handle, bottom[0]->count(), (const float *)bottom_data, 1, &maxIdx);
	// hipblasIsamin(handle, bottom[0]->count(), (const float *)bottom_data, 1, &minIdx);
	// hipblasDestroy(handle);

	// Dtype minVal = bottom[0]->cpu_data()[minIdx];
	// Dtype maxVal = bottom[0]->cpu_data()[maxIdx];
	// Dtype bin = (maxVal - minVal) / Dtype(pow(2, _bit_num) - 1.);
	// an array of minimums for calculation purpose
	Dtype* tmp_data = static_cast<Dtype*>(min_array_.mutable_cpu_data());	

	caffe_gpu_set(bottom[0]->count(), minVal, tmp_data);
	caffe_gpu_axpby(bottom[0]->count(), Dtype(1/bin), bottom_data, 
		- minVal / bin, tmp_data);
	caffe_gpu_set(bottom[0]->count(), Dtype(1.), top_data);
	// have to access cpu data for use of floor
	for (int i = 0; i < bottom[0]->count(); ++i) {
		tmp_data[i] = floor(tmp_data[i]);
	}
	LOG(INFO) << minVal;
	LOG(INFO) << maxVal;
	LOG(INFO) << bin;
	caffe_gpu_axpby(bottom[0]->count(), bin, tmp_data, minVal, top_data);
}
template<typename Dtype>
void QuantizationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
        const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    // DO NOTHING
}

INSTANTIATE_LAYER_GPU_FUNCS(QuantizationLayer);

}	// namespace caffe