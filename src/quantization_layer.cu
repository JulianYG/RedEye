#include "hip/hip_runtime.h"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/layers/quantization_layer.hpp"

namespace caffe {

template<typename Dtype>
void QuantizationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
	const vector<Blob<Dtype>*>& top) {

	const Dtype* bottom_data = bottom[0]->cpu_data();
	Dtype* top_data = top[0]->mutable_cpu_data();
	
	Dtype mx = bottom_data[0];
	Dtype mn = bottom_data[0];

	for (int i = 0; i < bottom[0]->count(); ++i) {
		if (mn > bottom_data[i]) {
			mn = bottom_data[i];
		} else if (mx < bottom_data[i]) {
			mx = bottom_data[i];
		}
	}
	Dtype bin = (mx - mn) / Dtype(pow(2, _bit_num) - 1.);
	for (int i = 0; i < top[0]->count(); ++i) {
		top_data[i] = bin * floor((bottom_data[i] - mn)/bin) + mn;
	}
}
template<typename Dtype>
void QuantizationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
        const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    // DO NOTHING
}

INSTANTIATE_LAYER_GPU_FUNCS(QuantizationLayer);

}	// namespace caffe